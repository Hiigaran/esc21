
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>
// Here you can set the device ID that was assigned to you
#define MYDEVICE 0


// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);
// Part 2 of 4: implement the kernel
__global__ void kernel( int *a, int dimx, int dimy ) {
    auto xindex = threadIdx.x + blockIdx.x * blockDim.x;
    auto yindex = threadIdx.y + blockIdx.y * blockDim.y;
    if(xindex < dimx && yindex < dimy){
       a[xindex * dimx + yindex] = xindex * dimx + yindex;
    }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main() {
    hipSetDevice(MYDEVICE);
// Part 1 and 4 of 4: set the dimensions of the matrix
    int dimx = 4;
    int dimy = 4;
    int num_bytes = dimx*dimy*sizeof(int);

    int *d_a=0, *h_a=0; // device and host pointers

    h_a = (int*)malloc(num_bytes);
    //allocate memory on the device
    hipMalloc(&d_a, num_bytes);

    if( NULL==h_a || NULL==d_a ) {
        std::cerr << "couldn't allocate memory" << std::endl;
        return 1;
    }

    // Part 2 of 4: define grid and block size and launch the kernel
    // NOTA: se le risorse assegnate sono minori della matrice devo
    // definire un secondo kernel
    dim3 grid, block;
    block.x = 5;
    block.y = 5;
    grid.x  = 10;
    grid.y  = 10;

    kernel<<<grid, block>>>( d_a, dimx, dimy );
    // block until the device has completed
    hipDeviceSynchronize();
    // check if kernel execution generated an error
    checkCUDAError("kernel execution");
    // device to host copy
    hipMemcpy(h_a, d_a, num_bytes, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy");
    // verify the data returned to the host is correct
    for(int row=0; row<dimx; row++)
    {
        for(int col=0; col<dimy; col++){
            std::cout << "index = " << row * dimx + col << std::endl;
            std::cout << "h_a = " << h_a[row * dimx + col] << std::endl;
            assert(h_a[row * dimx + col] == row * dimx + col);
        }
    }
    // free host memory
    free( h_a );
    // free device memory
    hipFree( d_a );

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    std::cout << "Correct!" << std::endl;

    return 0;
}


void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        std::cerr << "Cuda error: " << msg << " " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }                         
}
